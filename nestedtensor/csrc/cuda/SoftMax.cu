#include "hip/hip_runtime.h"
#include <nestedtensor/csrc/cuda/functions.h>

#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>
// #include <hip/hip_fp16.h>
// #include <hiprand/hiprand_kernel.h>
// #include <stdio.h>
// #include <stdlib.h>
#include <limits>
// #include <stdexcept>
#include <algorithm>

#define FINAL_MASK 0xffffffff
#define MAX_THREADS 1024
// Maximum sequence-length support based on the number of threads (2048) allowed
// in each block and this MAX is 8K For higher sequence length we need to use
// higher Max, like for 64K : 32
#define MAX_THREAD_ITERATIONS 8 // Maximum 8K
#define MAX_THREAD_STRIDE 32
#define MAX_WARP_NUM 32
#define THREADS 256
#define TILE_DIM 32
#define minus_infinity -1 * std::numeric_limits<float>::infinity()

#define WARP_SIZE 32

// #define CUDA_CHECK(callstr)                                                 \
//   {                                                                         \
//     hipError_t error_code = callstr;                                       \
//     if (error_code != hipSuccess) {                                        \
//       std::cerr << "CUDA error " << error_code << " at " << __FILE__ << ":" \
//                 << __LINE__;                                                \
//       assert(0);                                                            \
//     }                                                                       \
//   }

#define CUDA_1D_KERNEL_LOOP(i, n)                                 \
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)

#define CUDA_2D_KERNEL_LOOP(i, n, j, m)                             \
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (n);   \
       i += blockDim.x * gridDim.x)                                 \
    for (size_t j = blockIdx.y * blockDim.y + threadIdx.y; j < (m); \
         j += blockDim.y * gridDim.y)

#define DS_CUDA_NUM_THREADS 512
#define DS_MAXIMUM_NUM_BLOCKS 4096

inline int DS_GET_BLOCKS(const int N) {
  return std::max(
      std::min(
          (N + DS_CUDA_NUM_THREADS - 1) / DS_CUDA_NUM_THREADS,
          DS_MAXIMUM_NUM_BLOCKS),
      // Use at least 1 block, since CUDA does not allow empty block
      1);
}

// From deepspeed https://github.com/microsoft/DeepSpeed/blob/e549be607c0f85fc3eb91b3ce977f1d063d65f3c/csrc/transformer/softmax_kernels.cu

namespace cg = cooperative_groups;

// Fused attention + softmax
template <int tbSize, int blockStride, int tbSeq>
__global__ void attn_softmax(float* vals,
                             const float* attn_mask,
                             int heads,
                             int seq_length,
                             int iterations)
{
    __shared__ float partialSum[MAX_WARP_NUM];

    int warp_num = blockDim.x >> 5;

    int iteration_stride = blockDim.x;
    int block_width = blockStride * seq_length;

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<tbSize> g = cg::tiled_partition<tbSize>(b);

    int batch = blockIdx.x;
    int row = blockIdx.y;
    int max_threads_in_sequence = std::max(seq_length, tbSeq);
    int seq_lane = threadIdx.x % max_threads_in_sequence;

    int data_offset = batch * (gridDim.y * block_width) + row * block_width +
                      (threadIdx.x / max_threads_in_sequence) * seq_length;
    int mask_offset = batch * seq_length;

    int wid = threadIdx.x >> 5;
    int lane = threadIdx.x & 0x1f;

    float4* val_cast = reinterpret_cast<float4*>(vals);
    const float4* attn_mask_cast = reinterpret_cast<const float4*>(attn_mask);

    float4 data[MAX_THREAD_ITERATIONS];

    float max_val = minus_infinity;

    for (int i = 0; i < iterations; i++) {
        int data_id = i * iteration_stride + seq_lane;
        if (data_id < seq_length) {
            float4 mask = attn_mask_cast[mask_offset + data_id];
            data[i] = val_cast[data_offset + data_id];

            data[i].x += mask.x;
            data[i].y += mask.y;
            data[i].z += mask.z;
            data[i].w += mask.w;

            max_val = (data[i].x > max_val ? data[i].x : max_val);
            max_val = (data[i].y > max_val ? data[i].y : max_val);
            max_val = (data[i].z > max_val ? data[i].z : max_val);
            max_val = (data[i].w > max_val ? data[i].w : max_val);
        } else {
            data[i].x = minus_infinity;
            data[i].y = minus_infinity;
            data[i].z = minus_infinity;
            data[i].w = minus_infinity;
        }
    }

    for (int i = 1; i < tbSize; i *= 2) {
        auto temp = g.shfl_xor(max_val, i);
        max_val = (temp > max_val ? temp : max_val);
    }

    if (seq_length > tbSize) {
        if (lane == 0) partialSum[wid] = max_val;
        b.sync();

        if (lane < warp_num) max_val = partialSum[lane];

#ifndef __STOCHASTIC_MODE__
        b.sync();
#endif

        int iters = warp_num;
        if (seq_length < iteration_stride) iters = warp_num / (iteration_stride / seq_length);

        for (int i = 1; i < iters; i *= 2) {
            auto temp = g.shfl_xor(max_val, i);
            max_val = (temp > max_val ? temp : max_val);
        }

        max_val = g.shfl(max_val, threadIdx.x / tbSize);
    }

    float sum = 0;
    for (int i = 0; i < iterations; i++) {
        data[i].x = __expf(data[i].x - max_val);
        data[i].y = __expf(data[i].y - max_val);
        data[i].z = __expf(data[i].z - max_val);
        data[i].w = __expf(data[i].w - max_val);

        sum += (data[i].x + data[i].y + data[i].z + data[i].w);
    }

    for (int i = 1; i < tbSize; i *= 2) { sum += g.shfl_xor(sum, i); }

    if (seq_length > tbSize) {
        if (lane == 0) partialSum[wid] = sum;
        b.sync();

        if (lane < warp_num) sum = partialSum[lane];

#ifndef __STOCHASTIC_MODE__
        b.sync();
#endif

        int iters = warp_num;
        if (seq_length < iteration_stride) iters = warp_num / (iteration_stride / seq_length);

        for (int i = 1; i < iters; i *= 2) { sum += g.shfl_xor(sum, i); }

        sum = g.shfl(sum, threadIdx.x / tbSize);
    }

    sum += 1e-6;

    for (int i = 0; i < iterations; i++) {
        data[i].x /= sum;
        data[i].y /= sum;
        data[i].z /= sum;
        data[i].w /= sum;

        int data_id = i * iteration_stride + seq_lane;
        if (data_id < seq_length) val_cast[data_offset + data_id] = data[i];
    }
}

template <typename T>
void launch_attn_softmax(T*, const T*, int, int, int, hipStream_t, bool);

template <>
void launch_attn_softmax<float>(float* vals,
                                const float* attn_mask,
                                int batch_size,
                                int heads,
                                int sequence_length,
                                hipStream_t stream)
{
    const int threads = 128;
    int seq_length4 = sequence_length / 4;
    int seq2 = sequence_length * seq_length4;

    int block_compute_size =
        (seq_length4 < threads ? ((threads / seq_length4) * seq_length4) : seq_length4);
    dim3 grid_dim(batch_size, heads * seq2 / block_compute_size);

    int subblock_max_workload = MAX_THREAD_ITERATIONS * 4 * threads;

    dim3 block_dim(seq_length4 > threads ? ((sequence_length + subblock_max_workload - 1) /
                                            subblock_max_workload * threads)
                                         : threads);
    int iterations =
        (sequence_length < subblock_max_workload ? (seq_length4 + threads - 1) / threads
                                                 : MAX_THREAD_ITERATIONS);

    if (sequence_length <= 8)
        attn_softmax<2, (threads / 2), 2>
            <<<grid_dim, block_dim, 0, stream>>>(vals, attn_mask, heads, seq_length4, iterations);
    else if (sequence_length <= 16)
        attn_softmax<4, (threads / 4), 4>
            <<<grid_dim, block_dim, 0, stream>>>(vals, attn_mask, heads, seq_length4, iterations);
    else if (sequence_length <= 32)
        attn_softmax<8, (threads / 8), 8>
            <<<grid_dim, block_dim, 0, stream>>>(vals, attn_mask, heads, seq_length4, iterations);
    else if (sequence_length <= 64)
        attn_softmax<16, (threads / 16), 16>
            <<<grid_dim, block_dim, 0, stream>>>(vals, attn_mask, heads, seq_length4, iterations);
    else if (sequence_length <= 128)
        attn_softmax<32, (threads / 32), 32>
            <<<grid_dim, block_dim, 0, stream>>>(vals, attn_mask, heads, seq_length4, iterations);
    else if (sequence_length <= 256)
        attn_softmax<32, (threads / 64), 64>
            <<<grid_dim, block_dim, 0, stream>>>(vals, attn_mask, heads, seq_length4, iterations);
    else {
        const int threads = 256;
        block_compute_size =
            (seq_length4 < threads ? ((threads / seq_length4) * seq_length4) : seq_length4);
        dim3 grid_dim(batch_size, heads * seq2 / block_compute_size);

        int subblock_max_workload = MAX_THREAD_ITERATIONS * 4 * threads;

        dim3 block_dim(seq_length4 > threads ? ((sequence_length + subblock_max_workload - 1) /
                                                subblock_max_workload * threads)
                                             : threads);

        if (sequence_length <= 512)
            attn_softmax<32, (threads / 128), 128><<<grid_dim, block_dim, 0, stream>>>(
                vals, attn_mask, heads, seq_length4, iterations);
        else if (sequence_length < (MAX_THREADS * MAX_THREAD_ITERATIONS * 4))
            attn_softmax<32, 1, 128><<<grid_dim, block_dim, 0, stream>>>(
                vals, attn_mask, heads, seq_length4, iterations);
        //else
        //    exit(1);
        //    // throw std::runtime_error(
        //    //     "Unsupport Seq_Length! Check the restriction of the max_threads and "
        //    //     "max_thread_iterations!");
    }
}

template <typename T, int tbSize, int blockStride>
__global__ void softmax_backward_kernel(T* out_grad, const T* soft_inp, int seq_length)
{
    __shared__ float partialSum[MAX_WARP_NUM];

    int warp_num = blockDim.x >> 5;  // warp-count = num_threads / WARP_SIZE (32)

    int iteration_stride = blockDim.x;
    int block_width = blockStride * seq_length;

    int iterations = (seq_length < (MAX_THREAD_ITERATIONS * iteration_stride)
                          ? (seq_length + iteration_stride - 1) / iteration_stride
                          : MAX_THREAD_ITERATIONS);

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<tbSize> g = cg::tiled_partition<tbSize>(b);

    int row = blockIdx.x;
    int id = threadIdx.x;

    int wid = id >> 5;
    int lane = id & 0x1f;

    T val_reg[MAX_THREAD_ITERATIONS];
    T soft_reg[MAX_THREAD_ITERATIONS];
    float grad_reg = 0.0f;

#pragma unroll
    for (int i = 0; i < iterations; i++) {
        int data_id = i * iteration_stride + id;
        if (data_id < block_width) {
            val_reg[i] = out_grad[row * block_width + data_id];
            soft_reg[i] = soft_inp[row * block_width + data_id];

            grad_reg += ((float)val_reg[i] *
                         (float)soft_reg[i]);  // if done in half, the multiplication, we may lose
                                               // 2% of accuracy in computation!!
        }
    }
    for (int i = 1; i < tbSize; i *= 2) grad_reg += g.shfl_xor(grad_reg, i);

    if (seq_length > tbSize) {
        if (lane == 0) partialSum[wid] = grad_reg;
        b.sync();

        if (lane < warp_num) grad_reg = partialSum[lane];

        int iters = warp_num;
        if (seq_length < iteration_stride) iters = warp_num / (iteration_stride / seq_length);

        for (int i = 1; i < iters; i *= 2) grad_reg += g.shfl_xor(grad_reg, i);

        grad_reg = g.shfl(grad_reg, id / tbSize);
    }

    for (int i = 0; i < iterations; i++) {
        int data_id = i * iteration_stride + id;
        if (data_id < block_width) {
            float temp = (float)soft_reg[i] * ((float)val_reg[i] - grad_reg);
            out_grad[row * block_width + data_id] = (T)temp;
        }
    }
}

template <typename T, int ITERATIONS>
__global__ void softmax_backward_kernel_v2(T* grad /* input & output*/,
                                           const T* output,
                                           int softmax_length)
{
    int batch_idx = blockIdx.x * blockDim.y + threadIdx.y;
    int offset = batch_idx * softmax_length + threadIdx.x;

    grad += offset;
    output += offset;

    T grad_reg[ITERATIONS];
    T output_reg[ITERATIONS];
    float sum = 0.0;

#pragma unroll
    for (int i = 0; i < ITERATIONS; ++i) {
        int curr_idx = threadIdx.x + i * WARP_SIZE;
        if (curr_idx < softmax_length) {
            grad_reg[i] = grad[i * WARP_SIZE];
            output_reg[i] = output[i * WARP_SIZE];
            sum += (float)grad_reg[i] * (float)output_reg[i];
        }
    }

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    for (int i = 1; i < WARP_SIZE; i <<= 1) sum += g.shfl_xor(sum, i);

#pragma unroll
    for (int i = 0; i < ITERATIONS; ++i) {
        int curr_idx = threadIdx.x + i * WARP_SIZE;
        if (curr_idx < softmax_length)
            grad[i * WARP_SIZE] = (float)output_reg[i] * ((float)grad_reg[i] - sum);
    }
}

template <typename T>
void launch_attn_softmax_backward_v2(T* out_grad,
                                     const T* soft_inp,
                                     int batch_size,
                                     int heads,
                                     int seq_length,
                                     hipStream_t stream)
{
    // if ((seq_length % WARP_SIZE) != 0 || seq_length > 2048)
    //     throw std::runtime_error("Invalid sequence length found in softmax backward.");

    const int warps_per_block = 4;
    dim3 grid_dim(batch_size * heads * seq_length / warps_per_block);
    dim3 block_dim(WARP_SIZE, warps_per_block);

    switch (seq_length) {
        case 32:
            softmax_backward_kernel_v2<T, 1>
                <<<grid_dim, block_dim, 0, stream>>>(out_grad, soft_inp, seq_length);
            break;
        case 64:
            softmax_backward_kernel_v2<T, 2>
                <<<grid_dim, block_dim, 0, stream>>>(out_grad, soft_inp, seq_length);
            break;
        case 128:
            softmax_backward_kernel_v2<T, 4>
                <<<grid_dim, block_dim, 0, stream>>>(out_grad, soft_inp, seq_length);
            break;
        case 256:
            softmax_backward_kernel_v2<T, 8>
                <<<grid_dim, block_dim, 0, stream>>>(out_grad, soft_inp, seq_length);
            break;
        case 384:
            softmax_backward_kernel_v2<T, 12>
                <<<grid_dim, block_dim, 0, stream>>>(out_grad, soft_inp, seq_length);
            break;
        case 512:
            softmax_backward_kernel_v2<T, 16>
                <<<grid_dim, block_dim, 0, stream>>>(out_grad, soft_inp, seq_length);
            break;
        case 768:
            softmax_backward_kernel_v2<T, 24>
                <<<grid_dim, block_dim, 0, stream>>>(out_grad, soft_inp, seq_length);
            break;
        case 1024:
            softmax_backward_kernel_v2<T, 32>
                <<<grid_dim, block_dim, 0, stream>>>(out_grad, soft_inp, seq_length);
            break;
        case 2048:
            softmax_backward_kernel_v2<T, 64>
                <<<grid_dim, block_dim, 0, stream>>>(out_grad, soft_inp, seq_length);
            break;
        // default:
        //     throw std::runtime_error(
        //         std::string("Special sequence length found in softmax backward, seq_length: ") +
        //         std::to_string(seq_length));
    }
}

template void launch_attn_softmax_backward_v2<float>(float* out_grad,
                                                     const float* soft_inp,
                                                     int batch_size,
                                                     int heads,
                                                     int seq_length,
                                                     hipStream_t stream);



void launch_softmax_backward(
    at::Tensor vals,
    int batch_size,
    int heads,
    int sequence_length) {
    return vals;
}

void launch_softmax(
    at::Tensor vals,
    int batch_size,
    int heads,
    int sequence_length) {
    return vals;
}
